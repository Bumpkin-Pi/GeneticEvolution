#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <SDL2/SDL.h>
#include <thread>
#include <vector>

#define max_prey 100
#define max_predators 50
#define initial_prey 50
#define initial_predators 25
#define map_size_x 500
#define map_size_y 500
#define prey_max_speed 1
#define prey_max_turn_speed 1
#define predators_max_speed 1
#define predators_max_turn_speed 1
#define prey_view_distance 30
#define predators_view_distance 50

#define input_number 20
#define internal_number 5
#define output_number 2


using namespace std::literals;
using clock_type = std::chrono::high_resolution_clock;


class Network{
private:
    float input_to_internal_weight[internal_number][input_number];
    float internal_bias[internal_number];

    float internal_to_output_weight[output_number][internal_number];
    float output_bias[output_number];

    int internals[internal_number];
public:
    int inputs[input_number];
    int outputs[output_number];


    void think(){
        for (int n = 0; n < internal_number; n++){ // Input -> internals
            internals[n] = internal_bias[n];
            for (int i = 0; i < input_number; i++){
                internals[n] += inputs[i]*input_to_internal_weight[n][i]; // This feels like it might be the wrong way round
            }
            internals[n] = internals[n] / (1 + abs(internals[n]));
        }
        for (int n = 0; n < output_number; n++){ // internal -> outputs
            outputs[n] = output_bias[n];
            for (int i = 0; i < internal_number; i++){
                outputs[n] += internals[i]*internal_to_output_weight[n][i]; // This feels like it might be the wrong way round
            }
            outputs[n] = outputs[n] / (1 + abs(outputs[n]));
        }
    }
};


class Agent{
public:
    float pos[2];
    float direction;
    float energy;
    float age;

    Network network;

    Agent(float x, float y){
        this->pos[0] = x;
        this->pos[1] = y;
    }
    Agent(){
        this->pos[0] = 100;
        this->pos[1] = 100;
    }
};


const int screen_width = 1000;
const int screen_height = 1000;
SDL_Window* window = nullptr;
SDL_Renderer* renderer = nullptr;

bool running = true;
bool paused = false;


struct Data{
    Agent prey[max_prey];
    Agent predators[max_predators];
    int number_of_prey;
    int number_predators;
    int tick;
};
Data data;
Data *cuda_data;


void draw(){
    SDL_SetRenderDrawColor(renderer, 0,0,0,255);
    SDL_RenderClear(renderer);

    SDL_SetRenderDrawColor(renderer, 0,255,0,255);
    for (int n = 0; n < data.number_of_prey; n++){
        SDL_RenderDrawPoint(renderer, (int) data.prey[n].pos[0], (int) data.prey[n].pos[1]);
    }
    SDL_SetRenderDrawColor(renderer, 255,0,0,255);
    for (int n = 0; n < data.number_of_prey; n++){
        SDL_RenderDrawPoint(renderer, (int) data.predators[n].pos[0], (int) data.predators[n].pos[1]);
    }

    SDL_RenderPresent(renderer);
}






__global__ void prey_process(Data* data) {
    int index = (blockIdx.x); //TODO work for thread and block split


    return;
}
__global__ void predator_process(Data* data) {


    return;
}

void FixedUpdate(){ // Fixed time updater
    auto target_time = clock_type::now() + 30ms;
    while (running) {
        if (!paused){
            //TODO actual processing
            prey_process<<<32, 1>>>(cuda_data);
            hipDeviceSynchronize();
            hipMemcpy(&data, cuda_data, sizeof (data), hipMemcpyDeviceToHost);
        }
        std::this_thread::sleep_until(target_time);
        target_time += 30ms;
    }
}

int main(int argc, char **argv) {
    srand((unsigned) time(NULL));
    std::cout << "Genetic evolution!" << std::endl;

    SDL_Init(SDL_INIT_VIDEO);
    SDL_CreateWindowAndRenderer(screen_width, screen_height, 0, &window, &renderer);
    SDL_RenderSetScale(renderer,1,1);
    SDL_SetWindowTitle(window, "Genetic evolution");

    data.number_predators = initial_predators;
    data.number_of_prey = initial_prey;

    for (int n = 0; n < data.number_of_prey; n++){ //Scatter predators and prey
        data.prey[n].pos[0] = 1000*((float) rand())/(RAND_MAX);
        data.prey[n].pos[1] = 1000*((float) rand())/(RAND_MAX);
    }
    for (int n = 0; n < data.number_predators; n++){
        data.predators[n].pos[0] = 1000*((float) rand())/(RAND_MAX);
        data.predators[n].pos[1] = 1000*((float) rand())/(RAND_MAX);
    }


    hipMalloc(&cuda_data, sizeof (data));
    hipMemcpy(cuda_data, &data, sizeof (data), hipMemcpyHostToDevice);


    std::thread physicsThread(&FixedUpdate);

    while (running) {
        draw();
        SDL_Event event;
        while (SDL_PollEvent(&event)) {
            switch (event.type) {
                case SDL_QUIT:
                    running = false;
                    break;
                case SDL_KEYDOWN:
                    if (event.key.keysym.sym ==SDLK_SPACE)paused = !paused;
                    break;
            }
        }
    }

    physicsThread.join();



    return 0;
}
